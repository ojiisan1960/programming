/* Introduction to CUDA GPU programming
 *
 * https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
// __global__ tells the complier that this function
// runs on the GPU. In CUDA a function with __global__
// is known as a kernel
//
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  // Old memory allocation
  //float *x = new float[N];
  //float *y = new float[N];

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  //add(N, x, y);
  // launch with 1 GPU thread
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  //delete [] x;
  //delete [] y;
  hipFree(x);
  hipFree(y);

  return 0;
}
